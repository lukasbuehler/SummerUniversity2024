#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <iostream>
#include <cassert>

#include "util.hpp"

// a kernel that reverses a string of length n in place
__global__
void reverse_string(char* str, int n) {
    auto i = threadIdx.x;
    if(i < n/2) {
        const char tmp_char = str[i];
        str[i] = str[n - 1 - i];
        str[n - 1 - i] = tmp_char;
    }
}

int main(int argc, char** argv) {
    // check that the user has passed a string to reverse
    if(argc<2) {
        std::cout << "useage : ./string_reverse \"string to reverse\"\n" << std::endl;
        exit(0);
    }

    // determine the length of the string, and copy in to buffer
    auto n = strlen(argv[1]);
    auto string = malloc_managed<char>(n+1);
    std::copy(argv[1], argv[1]+n, string);
    string[n] = 0; // add null terminator

    std::cout << "string to reverse:\n" << string << "\n";

    // call the string reverse function
    assert(n <= 1024);
    reverse_string<<<1, n/2>>>(string, n);

    // print reversed string
    hipDeviceSynchronize();
    std::cout << "reversed string:\n" << string << "\n";

    // free memory
    hipFree(string);

    return 0;
}

